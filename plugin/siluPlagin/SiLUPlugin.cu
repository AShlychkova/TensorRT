#include "hip/hip_runtime.h"
#include "SiLUPlugin.h"
#include <hip/hip_fp16.h>

template <typename T_DATA>
__global__ void kernel(
    int N,
    T_DATA* inputs,
    T_DATA* outputs
)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < N){
        outputs[index] = inputs[index] / (1. + fexp(-inputs[index]);
    }
    __syncthreads();
}

template <typename T>
int SiLUInference(
    int batchSize,
    int iC,
    int iH,
    int iW,
    T* inputs,
    T* outputs,
    hipStream_t stream){
    const int nThreads = 512;
    int len = iC * iH * iW;

    int nBlocksCopy = (int)((float)len / nThreads) + 1;

    float stepACh = coordsRange / (float)(iH - 1);
    float stepACw = coordsRange / (float)(iW - 1);

    for(int i=0; i<batchSize; ++i){
        // NOTE: kernelCopy kernel can be replaced with hipMemcpy function
        kernel<<<nBlocksCopy, nThreads, 0, stream>>>(len, inputs, outputs);
        outputs += len;
    }

    hipError_t err = hipGetLastError();
    if ( hipSuccess != err )
    {
        fprintf( stderr, "cudaCheckError() failed at %s:%i : %s\n",
                 __FILE__, __LINE__, hipGetErrorString( err ) );
        return 1;
    }
    return 0;
}

int SiLUPlugin::enqueue(
    int batchSize, const void* const* inputs, void** outputs, void* workspace, hipStream_t stream)
{
    switch(iType){
    case DataType::kFLOAT:
        return SiLUInference(batchSize, iC, iH, iW (float*)inputs[0], (float*)outputs[0], stream);
    case DataType::kHALF:
        return SiLUInference(batchSize, iC, iH, iW, (__half*)inputs[0], (__half*)outputs[0], stream);
    }
    return 1;
}
